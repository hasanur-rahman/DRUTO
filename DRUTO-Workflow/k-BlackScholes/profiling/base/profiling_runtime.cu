#include <stdio.h>
#include <hip/hip_runtime.h>

const long long MAX_THREAD_NUMBER = 1000000;
__device__ unsigned long long counterArray[MAX_THREAD_NUMBER] = {0}; // this contains the execution count of a thread that is less than 10^6
__device__ unsigned long long counterArraySecond[MAX_THREAD_NUMBER] = {0}; // this contains the execution count of a thread that is greater than 10^6


extern "C" __device__ void bambooProfile(long bambooIndex) // is called when an instruction (bambooIndex) is executed
{
	int blockId = blockIdx.x 
		+ blockIdx.y * gridDim.x 
		+ gridDim.x * gridDim.y * blockIdx.z; 
	long long index = blockId * (blockDim.x * blockDim.y * blockDim.z)
		+ (threadIdx.z * (blockDim.x * blockDim.y))
		+ (threadIdx.y * blockDim.x)
		+ threadIdx.x; // index means threadIdx

	//printf("profiling: %lld\n", index);

	if (counterArray[MAX_THREAD_NUMBER - 1] < index) {
		counterArray[MAX_THREAD_NUMBER - 1] = index;
	}
	if (index < MAX_THREAD_NUMBER - 1LL)
		counterArray[index]++; // saving the current execution count of current thread
	else {
		if  (index < MAX_THREAD_NUMBER * 2 - 1LL)
			counterArraySecond[index - MAX_THREAD_NUMBER - 1LL]++; // saving the current execution count of current thread

	}
	//atomicAdd(&counterArray[index], 1ULL);
}
