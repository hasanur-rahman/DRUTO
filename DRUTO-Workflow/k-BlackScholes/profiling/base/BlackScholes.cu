#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */

//#include <helper_functions.h>  // helper functions for string parsing
//#include <hip/hip_runtime_api.h>  // helper functions CUDA error checking and initialization

#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <cstddef>
#include <time.h>

// Cuda Libraries
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "./header/profile_main.h"

// includes, kernels

#ifdef BAMBOO_PROFILING
#include "bamboo_profiling.cu"
#else
#include "bamboo_injection.cu"
#endif


////////////////////////////////////////////////////////////////////////////////
// Process an array of optN options on CPU
////////////////////////////////////////////////////////////////////////////////
extern "C" void BlackScholesCPU(float *h_CallResult, float *h_PutResult,
                                float *h_StockPrice, float *h_OptionStrike,
                                float *h_OptionYears, float Riskfree,
                                float Volatility, int optN);

////////////////////////////////////////////////////////////////////////////////
// Process an array of OptN options on GPU
////////////////////////////////////////////////////////////////////////////////
#include "BlackScholes_kernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
//float RandFloat(float low, float high) {
//  float t = (float)rand() / (float)RAND_MAX;
//  return (1.0f - t) * low + t * high;
//}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
//const int OPT_N = 4000000;
//int OPT_N = 0;
const int NUM_ITERATIONS = 512;
//int OPT_SZ = 0;
//const int OPT_SZ = OPT_N * sizeof(float);
const float RISKFREE = 0.02f;
const float VOLATILITY = 0.30f;

#define DIV_UP(a, b) (((a) + (b)-1) / (b))

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  // Start logs
  printf("[%s] - Starting...\n", argv[0]);

  if (argc != 2) {
      printf("Usage: Number of Options\n");
      exit(1);
  }

  const int OPT_N = atoi(argv[1]);
  const int OPT_SZ = OPT_N * sizeof(float); 

  //'h_' prefix - CPU (host) memory space
  float
      // Results calculated by CPU for reference
      *h_CallResultCPU,
      *h_PutResultCPU,
      // CPU copy of GPU results
      *h_CallResultGPU, *h_PutResultGPU,
      // CPU instance of input data
      *h_StockPrice, *h_OptionStrike, *h_OptionYears;

  //'d_' prefix - GPU (device) memory space
  float
      // Results calculated by GPU
      *d_CallResult,
      *d_PutResult,
      // GPU instance of input data
      *d_StockPrice, *d_OptionStrike, *d_OptionYears;

  double delta, ref, sum_delta, sum_ref, max_delta, L1norm; //, gpuTime;

  //StopWatchInterface *hTimer = NULL;
  int i;

  //findCudaDevice(argc, (const char **)argv);

  //sdkCreateTimer(&hTimer);

  printf("Initializing data...\n");
  printf("...allocating CPU memory for options.\n");
  h_CallResultCPU = (float *)malloc(OPT_SZ);
  h_PutResultCPU = (float *)malloc(OPT_SZ);
  h_CallResultGPU = (float *)malloc(OPT_SZ);
  h_PutResultGPU = (float *)malloc(OPT_SZ);
  h_StockPrice = (float *)malloc(OPT_SZ);
  h_OptionStrike = (float *)malloc(OPT_SZ);
  h_OptionYears = (float *)malloc(OPT_SZ);

  printf("...allocating GPU memory for options.\n");
  hipMalloc((void **)&d_CallResult, OPT_SZ); //checkCudaErrors(hipMalloc((void **)&d_CallResult, OPT_SZ));
  hipMalloc((void **)&d_PutResult, OPT_SZ); //checkCudaErrors(hipMalloc((void **)&d_PutResult, OPT_SZ));
  hipMalloc((void **)&d_StockPrice, OPT_SZ); //checkCudaErrors(hipMalloc((void **)&d_StockPrice, OPT_SZ));
  hipMalloc((void **)&d_OptionStrike, OPT_SZ); //checkCudaErrors(hipMalloc((void **)&d_OptionStrike, OPT_SZ));
  hipMalloc((void **)&d_OptionYears, OPT_SZ); //checkCudaErrors(hipMalloc((void **)&d_OptionYears, OPT_SZ));

  printf("...generating input data in CPU mem.\n");
  //sandd(time(NULL)); //5347);
  //srand(5347);

  // Generate options set
  for (i = 0; i < OPT_N; i++) {
    h_CallResultCPU[i] = 0.0f;
    h_PutResultCPU[i] = -1.0f;
    h_StockPrice[i] = (float) 1.0 * ((i+25) % 30); //RandFloat(5.0f, 30.0f);
    h_OptionStrike[i] = (float) 1.0 * ((i+55) % 100);//RandFloat(1.0f, 100.0f);
    h_OptionYears[i] = (float) std::max(0.25, 1.0 * ((i+3) % 10)); //RandFloat(0.25f, 10.0f);
  }
  
  // Copy options data to GPU memory for further processing
  //checkCudaErrors(
      hipMemcpy(d_StockPrice, h_StockPrice, OPT_SZ, hipMemcpyHostToDevice);// );
  //checkCudaErrors(
	hipMemcpy(d_OptionStrike, h_OptionStrike, OPT_SZ,
                             hipMemcpyHostToDevice); //);
  //checkCudaErrors(
      hipMemcpy(d_OptionYears, h_OptionYears, OPT_SZ, hipMemcpyHostToDevice); //);
  printf("Data init done.\n\n");

  printf("Executing Black-Scholes GPU kernel (%i iterations)...\n",
         NUM_ITERATIONS);
  hipDeviceSynchronize() ; //checkCudaErrors(hipDeviceSynchronize());
  //sdkResetTimer(&hTimer);
  //sdkStartTimer(&hTimer);
clock_t dkernel_time;
    double total_dkernel_time = 0.0;

  for (i = 0; i < NUM_ITERATIONS; i++) {
dkernel_time = clock();

    bambooLogKernelBegin(0);
    PROFILE(( 
    BlackScholesGPU<<<DIV_UP((OPT_N / 2), 128), 128 /*480, 128*/>>>(
        (float2 *)d_CallResult, (float2 *)d_PutResult, (float2 *)d_StockPrice,
        (float2 *)d_OptionStrike, (float2 *)d_OptionYears, RISKFREE, VOLATILITY,
        OPT_N)
    ));
    bambooLogKernelEnd(0); 
total_dkernel_time += ((double)(clock() - dkernel_time)) / CLOCKS_PER_SEC;

printf("dynamic kernel total exec time: %.6lf\n", total_dkernel_time);
    exit(0);
    //getLastCudaError("BlackScholesGPU() execution failed\n");
  }

  hipDeviceSynchronize() ; //checkCudaErrors(hipDeviceSynchronize());
  //sdkStopTimer(&hTimer);
  //gpuTime = sdkGetTimerValue(&hTimer) / NUM_ITERATIONS;

  // Both call and put is calculated
  printf("Options count             : %i     \n", 2 * OPT_N);
//  printf("BlackScholesGPU() time    : %f msec\n", gpuTime);
//  printf("Effective memory bandwidth: %f GB/s\n",
//         ((double)(5 * OPT_N * sizeof(float)) * 1E-9) / (gpuTime * 1E-3));
//  printf("Gigaoptions per second    : %f     \n\n",
//         ((double)(2 * OPT_N) * 1E-9) / (gpuTime * 1E-3));

    printf("BlackScholes, Size = %u options, NumDevsUsed = %u, Workgroup = %u\n", (2 * OPT_N), 1, 128);

//  printf(
//      "BlackScholes, Throughput = %.4f GOptions/s, Time = %.5f s, Size = %u "
//      "options, NumDevsUsed = %u, Workgroup = %u\n",
//      (((double)(2.0 * OPT_N) * 1.0E-9) / (gpuTime * 1.0E-3)), gpuTime * 1e-3,
//      (2 * OPT_N), 1, 128);

  printf("\nReading back GPU results...\n");
  // Read back GPU results to compare them to CPU results
  hipMemcpy(h_CallResultGPU, d_CallResult, OPT_SZ, hipMemcpyDeviceToHost); //checkCudaErrors(hipMemcpy(h_CallResultGPU, d_CallResult, OPT_SZ,
                             //hipMemcpyDeviceToHost));
  //checkCudaErrors(
      hipMemcpy(h_PutResultGPU, d_PutResult, OPT_SZ, hipMemcpyDeviceToHost); //);

  printf("Checking the results...\n");
  printf("...running CPU calculations.\n\n");
  // Calculate options values on CPU
  BlackScholesCPU(h_CallResultCPU, h_PutResultCPU, h_StockPrice, h_OptionStrike,
                  h_OptionYears, RISKFREE, VOLATILITY, OPT_N);

  printf("Comparing the results...\n");
  // Calculate max absolute difference and L1 distance
  // between CPU and GPU results
  sum_delta = 0;
  sum_ref = 0;
  max_delta = 0;

  FILE *fpo = fopen("output.txt", "w");
  for (i = 0; i < OPT_N; i++) {
    ref = h_CallResultCPU[i];
    delta = fabs(h_CallResultCPU[i] - h_CallResultGPU[i]);

    fprintf(fpo, "%.16f\n",delta);
    if (delta > max_delta) {
      max_delta = delta;
    }

    sum_delta += delta;
    sum_ref += fabs(ref);
  }
  fclose(fpo);


  L1norm = sum_delta / sum_ref;
  printf("L1 norm: %E\n", L1norm);

  fpo = fopen("output.txt", "a");
  fprintf(fpo, "sum_delta: %E\n", sum_delta);
  fprintf(fpo, "L1 norm: %E\n", L1norm);
  fprintf(fpo,"Max absolute error: %E\n", max_delta);
  fclose(fpo);
  printf("Max absolute error: %E\n\n", max_delta);

  printf("...releasing GPU memory.\n");
  hipFree(d_OptionYears); //checkCudaErrors(hipFree(d_OptionYears));
  hipFree(d_OptionStrike); //checkCudaErrors(hipFree(d_OptionStrike));
  hipFree(d_StockPrice); //checkCudaErrors(hipFree(d_StockPrice));
  hipFree(d_PutResult); //checkCudaErrors(hipFree(d_PutResult));
  hipFree(d_CallResult); //checkCudaErrors(hipFree(d_CallResult));

  printf("...releasing CPU memory.\n");
  free(h_OptionYears);
  free(h_OptionStrike);
  free(h_StockPrice);
  free(h_PutResultGPU);
  free(h_CallResultGPU);
  free(h_PutResultCPU);
  free(h_CallResultCPU);
  //sdkDeleteTimer(&hTimer);
  printf("Shutdown done.\n");

//  printf("\n[BlackScholes] - Test Summary\n");

//  if (L1norm > 1e-6) {
//    printf("Test failed!\n");
//    exit(EXIT_FAILURE);
//  }

//  printf(
//      "\nNOTE: The CUDA Samples are not meant for performance measurements. "
//      "Results may vary when GPU Boost is enabled.\n\n");
//  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}
