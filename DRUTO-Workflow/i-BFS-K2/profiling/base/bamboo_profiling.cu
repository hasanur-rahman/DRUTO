#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





const long long MAX_THREAD_NUMBER = 1000000;
extern "C" __device__ unsigned long long counterArray[MAX_THREAD_NUMBER], counterArraySecond[MAX_THREAD_NUMBER];

long long dynamicKernelIndex = 0;
int targetStaticKernelIndex = 1;

void bambooLogKernelBegin(int staticKernelIndex) {
	
}

void bambooLogKernelEnd(int staticKernelIndex) {

#ifdef KERNELTRACE
	cudaDeviceSynchronize();
#endif

	
	unsigned long long resultArray[MAX_THREAD_NUMBER] = {0};
	if (dynamicKernelIndex > 0) {
		exit(0);
	}

	if ((staticKernelIndex != targetStaticKernelIndex)){
		hipMemcpyToSymbol(HIP_SYMBOL(counterArray), &resultArray, MAX_THREAD_NUMBER * sizeof(unsigned long long), 0, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(counterArraySecond), &resultArray, MAX_THREAD_NUMBER * sizeof(unsigned long long), 0, hipMemcpyHostToDevice);
		return;
	}

	//unsigned long long secResultArray[MAX_THREAD_NUMBER] = {0};
	hipMemcpyFromSymbol(&resultArray, HIP_SYMBOL(counterArray), MAX_THREAD_NUMBER * sizeof(unsigned long long), 0, hipMemcpyDeviceToHost);
	//cudaMemcpyFromSymbol(&resultArraySec, counterArraySecond, MAX_THREAD_NUMBER * sizeof(unsigned long long), 0, cudaMemcpyDeviceToHost);	

	FILE *profileFile = fopen("Result/exec_count.txt", "a");
	fclose(profileFile);
	profileFile = fopen("Result/exec_count.txt", "a");
	for(long long i=0; i<MAX_THREAD_NUMBER-1LL; i++){
		if(resultArray[i] > 0){
			//printf(" -- index %lld -- counter %lld --\n", i, resultArray[i]);
			//FILE *profileFile = fopen("Result/exec_count.txt", "a");
			fprintf(profileFile, "%lld: %llu\n", i, resultArray[i]);
			//if (resultArraySec[i] > 0) 
			//	fprintf(profileFile, "%lld: %llu\n", i+MAX_THREAD_NUMBER, resultArraySec[i]);
			//fclose(profileFile);
		}
		
	}
	fclose(profileFile);

	
	if (resultArray[MAX_THREAD_NUMBER-1LL] > MAX_THREAD_NUMBER-2LL){
		profileFile = fopen("Result/exec_count.txt", "a");
		memset(resultArray, 0, sizeof(resultArray));
		hipMemcpyFromSymbol(&resultArray, HIP_SYMBOL(counterArraySecond), MAX_THREAD_NUMBER * sizeof(unsigned long long), 0, hipMemcpyDeviceToHost);
		for(long long i=0; i<MAX_THREAD_NUMBER; i++){
			if(resultArray[i] > 0){
				//printf(" -- index %lld -- counter %lld --\n", i, resultArray[i]);
				//FILE *profileFile = fopen("Result/exec_count.txt", "a");
				fprintf(profileFile, "%lld: %llu\n", i + MAX_THREAD_NUMBER - 1LL, resultArray[i]);
				//if (resultArraySec[i] > 0) 
				//	fprintf(profileFile, "%lld: %llu\n", i+MAX_THREAD_NUMBER, resultArraySec[i]);
				//fclose(profileFile);
			}
			
		}
		fclose(profileFile);
	}

	
//	cudaMemcpyToSymbol(counterArray, &resultArray, MAX_THREAD_NUMBER * sizeof(unsigned long long), 0, cudaMemcpyHostToDevice);
	dynamicKernelIndex++;
	
}
