#include <stdio.h>
#include <hip/hip_runtime.h>

const long long MAX_THREAD_NUMBER = 1000000;
__device__ unsigned long long counterArray[MAX_THREAD_NUMBER] = {0};
__device__ unsigned long long counterArraySecond[MAX_THREAD_NUMBER] = {0};


extern "C" __device__ void bambooProfile(long bambooIndex)
{
	int blockId = blockIdx.x 
		+ blockIdx.y * gridDim.x 
		+ gridDim.x * gridDim.y * blockIdx.z; 
	long long index = blockId * (blockDim.x * blockDim.y * blockDim.z)
		+ (threadIdx.z * (blockDim.x * blockDim.y))
		+ (threadIdx.y * blockDim.x)
		+ threadIdx.x;

	//printf("profiling: %lld\n", index);

	if (counterArray[MAX_THREAD_NUMBER - 1] < index) {
		counterArray[MAX_THREAD_NUMBER - 1] = index;
	}
	if (index < MAX_THREAD_NUMBER - 1LL)
		counterArray[index]++;
	else {
		if  (index < MAX_THREAD_NUMBER * 2 - 1LL)
			counterArraySecond[index - MAX_THREAD_NUMBER - 1LL]++;

	}
	//atomicAdd(&counterArray[index], 1ULL);
}
